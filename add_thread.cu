
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void myKernelAdd(int *a, int *b, int n) {

    int index = threadIdx.x;
    int stride = blockDim.x;

    for(int i = index; i < n; i+= stride) {
        a[i] = a[i] + b[i];
        if (i%blockDim.x==0) {
            printf("element #%d = %d, on block %d, thread %d, blockDim x y z %d %d %d\n",
            i, a[i], blockIdx.x, threadIdx.x, blockDim.x, blockDim.y, blockDim.z);
        }
    }
}

int main() {
    int N = 512;
    int *a, *b;

    // allocate unified memory, memory accessible from CPU and GPU
    hipMallocManaged(&a, N*sizeof(int));
    hipMallocManaged(&b, N*sizeof(int));

    for (int i = 0; i < N ; i++) {
        a[i] = i;
        b[i] = 0;
    }

    myKernelAdd<<<1,128>>>(a, b, N);

    hipDeviceSynchronize();
    hipFree(a);
    hipFree(b);

    return 0;
}