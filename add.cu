
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void myKernelAdd(int *a, int *b, int n) {

    for(int i = 0; i < n; i++) {
        a[i] = a[i] + b[i];
        printf("element #%d = %d, on block %d, thread %d\n", i, a[i], blockIdx.x, threadIdx.x);
    }
}

int main() {
    int N = 64;
    int *a, *b;

    // allocate unified memory, memory accessible from CPU and GPU
    hipMallocManaged(&a, N*sizeof(int));
    hipMallocManaged(&b, N*sizeof(int));

    for (int i = 0; i < N ; i++) {
        a[i] = i;
        b[i] = 0;
    }

    myKernelAdd<<<1,32>>>(a, b, N);

    hipDeviceSynchronize();
    hipFree(a);
    hipFree(b);

    return 0;
}