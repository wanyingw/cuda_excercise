
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void myKernelAdd(int *a, int *b, int n) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < n; i+= stride) {
        a[i] = a[i] + b[i];
        if (threadIdx.x==0) {
            printf("element #%d = %d, on block %d, thread %d, blockDim x y z %d %d %d\n",
            i, a[i], blockIdx.x, threadIdx.x, blockDim.x, blockDim.y, blockDim.z);
        }
    }
}

int main() {
    int N = 1024;
    int *a = nullptr;
    int *b = nullptr;
    int numThreads = 256; // number of threads per block
    int numBlocks = (N + numThreads - 1) / numThreads;

    // allocate unified memory, memory accessible from CPU and GPU
    hipMallocManaged(&a, N*sizeof(int));
    hipMallocManaged(&b, N*sizeof(int));

    for (int i = 0; i < N ; i++) {
        a[i] = i;
        b[i] = 0;
    }

    myKernelAdd<<<numBlocks, numThreads>>>(a, b, N);

    hipDeviceSynchronize();
    hipFree(a);
    hipFree(b);

    return 0;
}