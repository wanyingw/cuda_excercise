
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__ void myKernel(float f) {
    if (threadIdx.x == 0) {
        printf("hello from gpu block %d, thread %d, f=%f\n", blockIdx.x, threadIdx.x, f);}
}

int main() {
    size_t printfBufferSz;
    hipDeviceGetLimit(&printfBufferSz, hipLimitPrintfFifoSize);
    std::cout << printfBufferSz << std::endl;
    hipDeviceSetLimit(hipLimitPrintfFifoSize, 10000);
    hipDeviceGetLimit(&printfBufferSz, hipLimitPrintfFifoSize);
    std::cout << printfBufferSz << std::endl;

    myKernel<<<5,5>>>(2.333f);
    hipDeviceSynchronize();
    printf("hello from cpu\n");
    return 0;
}